#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>
#include "utils.h"

#define BLOCKDIM 1024 // Must be power of 2

__device__
float maxKer(float a, float b){
	if(a > b) return a;
	else return b;
}

__device__
float minKer(float a, float b){
	if(a < b) return a;
	else return b;
}

__global__
void reduce_maxmin(const float * const vec, int vecSize, float * const max, float * const min, int * const lock){
	// We will use 1D block organization and 1D thread organization
	// For each block, we calculate the range in the vector it has to work with
	// Then we reduce it
	// Then we reduce all blocks
	// Any vector element out-of-range gets the --identity-- value
	// TODO: We will waste half of the threads allocated for now. Fix this later.

	// DOC
	// - number of threads must be multiple of 2
	// - total number of threads must exceed the vector size
	// - 'max' must be initialized to a low value (identity for max)
	// - 'min' must be initialized to a high value (identity for min)
	// - 'lock' must be initialized to 0

	int beg = blockIdx.x * blockDim.x;
	int size = blockDim.x;

	// Get shared memory (intermediate values)
	// 'size' is expected to be at most 1024, so we're allocating at most 4kB of shared memory
	__shared__ float maxVec[BLOCKDIM/2];
	__shared__ float minVec[BLOCKDIM/2];

	// Copy elements from vec to shared memory, while doing the first iteration of the reduce
	int step = size >> 1;
	if(threadIdx.x < step){
		int right, left;
		right = beg + threadIdx.x + step;
		left = beg + threadIdx.x;

		// Elements out of bounds receive an identity value.
		if(left >= vecSize){
			maxVec[threadIdx.x] = *min;
			minVec[threadIdx.x] = *max;
		} else if(right >= vecSize){
			maxVec[threadIdx.x] = vec[left];
			minVec[threadIdx.x] = vec[left];
		} else {
			maxVec[threadIdx.x] = maxKer(vec[beg + threadIdx.x], vec[beg + threadIdx.x + step]);
			minVec[threadIdx.x] = minKer(vec[beg + threadIdx.x], vec[beg + threadIdx.x + step]);
		}
	}

	__syncthreads();

	// Reduce stuff in shared memory
	for(step >>= 1; step > 0; step >>= 1){
		if(threadIdx.x < step){
			if(maxVec[threadIdx.x + step] > maxVec[threadIdx.x])
				maxVec[threadIdx.x] = maxVec[threadIdx.x + step];

			if(minVec[threadIdx.x + step] < minVec[threadIdx.x])
				minVec[threadIdx.x] = minVec[threadIdx.x + step];
		}
	}

	if(threadIdx.x == 0){
		while( atomicCAS(lock, 0, 1) != 0 );
		*max = maxKer(*max, maxVec[0]);
		*min = minKer(*min, minVec[0]);
		*lock = 0;
	}
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
	int vecSize = numRows * numCols;
	int numBlocks = vecSize / (double) BLOCKDIM + 1;

	/*
	float *vector = (float *) malloc(sizeof(float) * vecSize);
	hipMemcpy(vector, d_logLuminance, sizeof(float) * vecSize, hipMemcpyDeviceToHost);
	float max = 0, min = 275;
	for(int i = 0; i < vecSize; i++){
		max = std::max(max, vector[i]);
		min = std::min(min, vector[i]);
	}
	printf("%f %f\n", min, max);
	free(vector);
	*/
	// Max should be 2.189105
	// Min should be -4

	// Get device memory
	float *d_max, *d_min;
	int *d_lock;

	checkCudaErrors(hipMalloc(&d_max, sizeof(float)));
	checkCudaErrors(hipMalloc(&d_min, sizeof(float)));
	checkCudaErrors(hipMalloc(&d_lock, sizeof(int)));
	checkCudaErrors(hipMemset(d_max, -275, sizeof(float)));
	checkCudaErrors(hipMemset(d_min, 275, sizeof(float)));
	checkCudaErrors(hipMemset(d_lock, 0, sizeof(int)));

	reduce_maxmin<<<numBlocks, BLOCKDIM>>>(d_logLuminance, vecSize, d_max, d_min, d_lock);
	checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_max));
	checkCudaErrors(hipFree(d_min));
	checkCudaErrors(hipFree(d_lock));

	printf("Min: %f, Max: %f\n", min_logLum, max_logLum);

	//TODO
	/*
		1) find the minimum and maximum value in the input logLuminance channel
			 store in min_logLum and max_logLum
		2) subtract them to find the range
		3) generate a histogram of all the values in the logLuminance channel using
			 the formula: bin = (lum[i] - lumMin) / lumRange * numBins
		4) Perform an exclusive scan (prefix sum) on the histogram to get
			 the cumulative distribution of luminance values (this should go in the
			 incoming d_cdf pointer which already has been allocated for you)
	*/
}
