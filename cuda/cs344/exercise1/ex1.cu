#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.h"
#include "utils.h"

const int N= 1024;		// matrix size is NxN
const int K= 1;	    	// TODO, set K to the correct value and tile size will be KxK


// to be launched with one thread per element, in KxK threadblocks
// thread (x,y) in grid writes element (i,j) of output matrix 
__global__ void 
transpose_parallel_per_element(float in[], float out[])
{
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	out[col*N + row] = in[row*N + col];
}

//The following functions and kernels are for your reference
void 
transpose_CPU(float in[], float out[])
{
	for(int j=0; j < N; j++)
    	for(int i=0; i < N; i++)
      		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched on a single thread
__global__ void 
transpose_serial(float in[], float out[])
{
	for(int j=0; j < N; j++)
		for(int i=0; i < N; i++)
			out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per row of output matrix
__global__ void 
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x;

	for(int j=0; j < N; j++)
		out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

void fill_matrix(float *in, int N){
	for(int i = 0; i < N*N; i++)
		in[i] = i;
}

bool compare_matrices(float *a, float *b, int N){
	for(int i = 0; i < N*N; i++)
		if(a[i] != b[i]) return true;
	return false;
}

void print_matrix(float *a, int N){
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			printf("%d ", (int) a[i*N + j]);
		}
		printf("\n");
	}
}

int main(int argc, char **argv)
{
	int numbytes = N * N * sizeof(float);

	float *in = (float *) malloc(numbytes);
	float *out = (float *) malloc(numbytes);
	float *gold = (float *) malloc(numbytes);

	fill_matrix(in, N);
	transpose_CPU(in, gold);

	float *d_in, *d_out;

	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

	GpuTimer timer;

	timer.Start();
	transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_row: %g ms.\nVerifying transpose...%s\n",
		   timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");
	
/*  
 * Now time each kernel and verify that it produces the correct result.
 *
 * To be really careful about benchmarking purposes, we should run every kernel once
 * to "warm" the system and avoid any compilation or code-caching effects, then run 
 * every kernel 10 or 100 times and average the timings to smooth out any variance. 
 * But this makes for messy code and our goal is teaching, not detailed benchmarking.
 */

	dim3 blocks(N/16, N/16);
	dim3 threads(16, 16);

	timer.Start();
	transpose_parallel_per_element<<<blocks,threads>>>(d_in, d_out);
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element: %g ms.\nVerifying transpose...%s\n",
		   timer.Elapsed(), compare_matrices(out, gold, N) ? "Failed" : "Success");

	hipFree(d_in);
	hipFree(d_out);
}
