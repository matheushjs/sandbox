#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <blelloch_scan.h>

// pow2(X) returns 2^X
#define pow2(X) (1 << X)

// Step should be 1, 2, 3... up to log2(vecSize) (round up)
// That's because the number of steps required is known once you have the size of the vector
// Also, the current step value determines what each thread will do at each point
// Number of threads required per step:
//    step1:   vecSize/2
//    step2:   vecSize/4
// and so on.
__global__
void blelloch_reduce(unsigned int *d_vec, int vecSize, int step){
	const int myIdx = blockDim.x * blockIdx.x + threadIdx.x;
	const int myElem = pow2(step) * (myIdx + 1) - 1;
	const int otherElem = myElem - pow2(step - 1);

	if(myElem >= vecSize) return;
	d_vec[myElem] = d_vec[myElem] OPERATOR d_vec[otherElem];
}

// Be nSteps = log2(vecSize) (round up)
// Step should be nSteps, nSteps-1, nSteps-2, ..., 1
// That's because the downsweep element accessing pattern is the opposite of the reduce pattern
// Number of threads required per step:
//    step nStep:   vecSize/pow(nStep)
//    step nStep-1: vecSize/pow(nStep-1)
//    ...
//    step2:   vecSize/4
//    step1:   vecSize/2
__global__
void blelloch_downsweep(unsigned int *d_vec, int vecSize, int step){
	const int myIdx = blockDim.x * blockIdx.x + threadIdx.x;
	const int myElem = pow2(step) * (myIdx + 1) - 1;
	const int otherElem = myElem - pow2(step - 1);

	if(myElem >= vecSize) return;

	int aux = d_vec[myElem];
	d_vec[myElem] = d_vec[myElem] OPERATOR d_vec[otherElem];
	d_vec[otherElem] = aux;
}

// Size of d_vec must be a power of 2.
void xscan(unsigned int *d_vec, int vecSize){
	int size = 1;
	int nSteps = 0;
	while(size < vecSize){
		size <<= 1; // Lowest power of 2 greater than or equal to vecSize
		nSteps++;   // The power itself
	}

	// First reduce
	for(int step = 1; step <= nSteps; step++){
		const int thrCount = vecSize / pow2(step);
		const int thrPerBlock = 256; //Reasonable number of threads in a block
		const int nBlocks = thrCount / thrPerBlock + 1;

		blelloch_reduce<<<nBlocks, thrPerBlock>>>(d_vec, size, step);
	}

	// Put identity value into last element
	unsigned int aux = IDENTITY;
	hipMemcpy( &d_vec[vecSize - 1], &aux, sizeof(int), hipMemcpyHostToDevice);

	// Now downsweep
	for(int step = nSteps; step >= 1; step--){
		const int thrCount = vecSize / pow2(step);
		const int thrPerBlock = 256; //Reasonable number of threads in a block
		const int nBlocks = thrCount / thrPerBlock + 1;

		blelloch_downsweep<<<nBlocks, thrPerBlock>>>(d_vec, size, step);
	}
}
